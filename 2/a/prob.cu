#include<iostream>
#include<fstream>
#include<strstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include<stdio.h>
#include<stdlib.h>

using namespace std;

int min2(int,int);
int min1(int,int,int);



__global__ void initialisation(int* G1, int* G2, int* D1, int* I2, int ge, int gi, int n)
{
    	int j = threadIdx.x;
    	if(j>0)
    	{
    		G1[j]=gi+ge*j;
       		D1[j]=G1[j]+ge;
	}
        if(j==0)
        {
		G1[0]=0;
    		G2[0]=gi+ge*n;
               	I2[0]=G2[0]+ge;
     	}

}

__global__ void copy_array(float* A, float* B)
{
    int i = threadIdx.x;
    B[i] = A[i];
}

int main(int argc, char** argv)
{
	int i,j,n;
	
	char buffer1[2048];								//--- taking input from file
	char buffer2[2048];								//--- n,s1,s2 are the corresponding three lines of input
	char buffer3[2048];
	istrstream str1(buffer1, 2048);
	istrstream str2(buffer2, 2048);
	istrstream str3(buffer3, 2048);
	ifstream indata("rand-1024-in.txt");		
	indata.getline(buffer1, 2048);
	indata.getline(buffer2, 2048);
	indata.getline(buffer3, 2048);
	str1>>n;

	size_t size = (n) * sizeof(char);
	size_t size1 = (n)*sizeof(int);

	char* h_s1 = (char*)malloc(size);
	char* h_s2 = (char*)malloc(size);

	for(i=0;i<n;i++)
	{
		str2>>h_s1[i];
		str3>>h_s2[i];
	}												//--- end of input
		
	
	int* h_D1 = (int*)malloc(size1);
	int* h_D2 = (int*)malloc(size1);
	int* h_G1 = (int*)malloc(size1);
	int* h_G2 = (int*)malloc(size1);
	int* h_I1 = (int*)malloc(size1);
	int* h_I2 = (int*)malloc(size1);
	int* h_u = (int*)malloc(size1);
	int* h_v = (int*)malloc(size1);
	int* h_se = (int*)malloc(size1);


//	int h_D1[n], h_D2[n],h_G1[n], h_G2[n],h_I1[n+1],h_I2[n+1],h_u[n],h_v[n],h_se[n];	
	int gi,ge,s,cost;
	gi=2;ge=1;

	int *d_D1, *d_D2;
    	hipMalloc(&d_D1, size1);
	hipMalloc(&d_D2, size1);
    	int *d_G1, *d_G2;
    	hipMalloc(&d_G1, size1);
	hipMalloc(&d_G2, size1);
    	int *d_I1, *d_I2;
    	hipMalloc(&d_I1, size1);
	hipMalloc(&d_I2, size1);
    	int *d_u, *d_v, *d_se;
    	hipMalloc(&d_u, size1);
	hipMalloc(&d_v, size1);
	hipMalloc(&d_se, size1);
    
/*	char *d_s1, *d_s2;
	cudaMalloc(&d_s1, size);
	cudaMalloc(&d_s2, size);

    	cudaMemcpy(d_s1, h_s1, size, cudaMemcpyHostToDevice);
	cudaMemcpy(d_s2, h_s2, size, cudaMemcpyHostToDevice);
*/
	
//	d_G1[0]=0;
	initialisation<<<1, n>>>(d_G1, d_G2, d_D1, d_I2, gi, ge, n);



/*	h_G1[0]=0;										
		for(j=0;j<n;j++)
		{
			if(j>0)
			{
				h_G1[j]=gi+ge*j;
				h_D1[j]=h_G1[j]+ge;
			}
			if(j==0)
			{
				h_G2[0]=gi+ge*i;
				h_I2[0]=h_G2[0]+ge;
			}
		}
*/


	hipMemcpy(h_D1, d_D1, size1, hipMemcpyDeviceToHost);
    	hipMemcpy(h_G1, d_G1, size1, hipMemcpyDeviceToHost);
   	hipMemcpy(h_G2, d_G2, size1, hipMemcpyDeviceToHost);
	hipMemcpy(h_I2, d_I2, size1, hipMemcpyDeviceToHost);	
	
	for(i=1;i<n;i++)								//--- calculation of D,I,G
	{
		for(j=1;j<n;j++)
		{
			
			if(i>0 && j>0)
			{
				h_D2[j]=min(h_D1[j],h_G1[j]+gi)+ge;
				
				if(h_s1[i]!=h_s2[j])
					s=1;
				else
					s=0;
				
				h_u[j] = min(h_D2[j],h_G1[j-1]+s);
				
				h_v[j] = h_u[j]+gi-j*ge;
				
				h_G2[j]=min(h_u[j],h_I2[j]);
				
				h_se[j]=h_v[j];
				for(int k=1; k<=j;k++)
				{
					if(h_se[j] > h_v[k])
						h_se[j]=h_v[k];
				}
				
				h_I2[j+1]=h_se[j]+(j+1)*ge;
				if(j==1)
					h_I2[1]=min(h_I2[0],h_G2[0]+gi) + ge;
		
				h_G2[j]=min1(h_D2[j],h_I2[j],h_G1[j-1]+s);
			}
		}
	
	
	

		for(int t=0;t<n;t++)
		{
			h_D1[t]=h_D2[t];
			h_I1[t]=h_I2[t];
			h_G1[t]=h_G2[t];
		}
	
	}													//--- end of loop for calculation
	
	cost=min1(h_D2[n-1],h_I2[n-1],h_G2[n-1]);			//--- allignment cost
	
	cout<<"Optimal Allignment cost: "<<cost<<endl;
	
	return 0;
}
int min2(int a,int b)
{
	if(a>b)
		return b;
	else
		return a;
}

int min1(int a,int b,int c)
{
	int m=a;
    if(m>b)
		m=b;
    if(m>c)
		m=c;
    return m;
}
