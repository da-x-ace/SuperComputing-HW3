#include<iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include<stdio.h>
#include<stdlib.h>
#include <sys/types.h>
#include <time.h>



using namespace std;

__device__ int min2(int a, int b)
{
	int m = a;
	if(m > b)
		m=b;
	return m;
}
__device__ int min1(int a,int b,int c)
{
	int m=a;
    if(m>b)
		m=b;
    if(m>c)
		m=c;
    return m;
}	

int min3(int a,int b,int c)
{
	int m=a;
    if(m>b)
		m=b;
    if(m>c)
		m=c;
    return m;
}	

void swapPointers(int **prev,int **current)
{
     int * temp = *prev;
     *prev      = *current;
     *current   = temp;
}



__global__ void main_work(char* s1, char *s2, int* G1, int* G2, int* D1, int* D2, int* I1, int* I2, int* u, int* v, int* se, int ge, int gi, int n, int i, int numElements)
{
	int j =   blockIdx.x * blockDim.x + threadIdx.x;
	int s=0;
	
	if( j > numElements ) return ;
	
	__syncthreads();
			if(j==0 && i>0)
			{
				G2[0]=gi+ge*i;
				I2[0]=G2[0]+ge;
			}
			if(j>0 && i == 0)
			{
				G1[j]=gi+ge*j;
				D1[j]=G1[j]+ge;
			}
			if(i>0 && j>0)
			{
				D2[j]=min2(D1[j],G1[j]+gi)+ge;
				__syncthreads();
				if(s1[i]!=s2[j])
					s=1;
				else
					s=0;
				__syncthreads();
				u[j] = min2(D2[j],G1[j-1]+s);
				__syncthreads();
				v[j] = u[j]+gi-j*ge;
				__syncthreads();
				se[j]=v[j];
				__syncthreads();
				for(int k=1; k<=j;k++)
				{
					if(se[j] > v[k])
						se[j]=v[k];
				}
				__syncthreads();
				I2[j+1]=se[j]+(j+1)*ge;
				__syncthreads();
				if(j==1)
					I2[1]=min2(I2[0],G2[0]+gi) + ge;
				__syncthreads();
				G2[j]=min1(D2[j],I2[j],G1[j-1]+s);
			}
}


int main(int argc, char** argv)
{
	int i,n;
	time_t t0, t1;
    clock_t c0,c1;
	char skip;

	scanf("%d",&n);
	printf("%d \n",n);
	while(1)
	{
		scanf("%c",&skip);
		if(skip == '\n')
			break;
	}
	size_t size = (n+1) * sizeof(char);
	size_t size1 = (n+1)*sizeof(int);

	char* h_s1 = (char*)malloc(size);
	char* h_s2 = (char*)malloc(size);
	
	for(i=1; i<=n; i++)
	{
		scanf("%c",&h_s1[i]);
	}
	while(1)
	{
		scanf("%c",&skip);
		if(skip == '\n')
			break;
	}
	for(i=1; i<=n; i++)
	{
		scanf("%c",&h_s2[i]);
	}
	

	t0=time(NULL);
        c0=clock();

        printf ("\tbegin (wall):            %ld\n", (long) t0);
        printf ("\tbegin (CPU):             %d\n", (int) c0);
	
	int* h_D1 = (int*)malloc(size1);
	int* h_D2 = (int*)malloc(size1);
	int* h_G1 = (int*)malloc(size1);
	int* h_G2 = (int*)malloc(size1);
	int* h_I1 = (int*)malloc(size1);
	int* h_I2 = (int*)malloc(size1);
	
	int gi,ge,cost;
	gi=2;ge=1;
	
		int *d_D1, *d_D2;
    	hipMalloc(&d_D1, size1);
		hipMalloc(&d_D2, size1);
    	int *d_G1, *d_G2;
    	hipMalloc(&d_G1, size1);
		hipMalloc(&d_G2, size1);
    	int *d_I1, *d_I2;
    	hipMalloc(&d_I1, size1);
		hipMalloc(&d_I2, size1);
    	int *d_u, *d_v, *d_se;
    	hipMalloc(&d_u, size1);
		hipMalloc(&d_v, size1);
		hipMalloc(&d_se, size1);
	
	char *d_s1, *d_s2;
	hipMalloc(&d_s1, size);
	hipMalloc(&d_s2, size);

    hipMemcpy(d_s1, h_s1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_s2, h_s2, size, hipMemcpyHostToDevice);
	
	h_G1[0]=0;										
	hipMemcpy(d_G1, h_G1, size1, hipMemcpyHostToDevice);
	
	int num_threads;
	int num_blocks;
	int numElements;
	
		numElements = n+1;
		if( numElements <= 256) {
		 num_threads = numElements;
		 num_blocks =1 ;
		 }
		 else
		 {
		   num_threads = 256 ;
		   num_blocks = numElements/num_threads + ((numElements/num_threads == 0) ? 0 : 1) ;
		 }

	for(i=0;i<=n;i++)								
	{
	
		
		main_work<<<num_blocks,num_threads>>>(d_s1, d_s2, d_G1, d_G2, d_D1, d_D2, d_I1, d_I2, d_u, d_v, d_se, ge, gi, n, i, numElements);
		
		if(i > 0)
		{
			swapPointers(&d_D1,&d_D2);
			swapPointers(&d_I1,&d_I2);
			swapPointers(&d_G1,&d_G2);
		}

	}													//--- end of loop for calculation
	
	hipMemcpy(h_D1, d_D1, size1, hipMemcpyDeviceToHost);
	hipMemcpy(h_D2, d_D2, size1, hipMemcpyDeviceToHost);
    	hipMemcpy(h_G1, d_G1, size1, hipMemcpyDeviceToHost);
   	hipMemcpy(h_G2, d_G2, size1, hipMemcpyDeviceToHost);
	hipMemcpy(h_I1, d_I1, size1, hipMemcpyDeviceToHost);
	hipMemcpy(h_I2, d_I2, size1, hipMemcpyDeviceToHost);
	
	cost=min3(h_D1[n],h_I1[n],h_G1[n]);			//--- allignment cost
	
	cout<<"Optimal Allignment cost: "<<cost<<endl;
	
	t1=time(NULL);
        c1=clock();
        printf ("\telapsed wall clock time: %ld\n", (long) (t1 - t0));
        printf ("\telapsed CPU time:        %f\n", (float) (c1 - c0)/CLOCKS_PER_SEC);
	
	hipFree(d_D1);
	hipFree(d_D2);
	hipFree(d_I1);
	hipFree(d_I2);
	hipFree(d_G1);
	hipFree(d_G2);
	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_se);
	
	free(h_s1);
	free(h_s2);
	return 0;
}

