#include<iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include<stdio.h>
#include<stdlib.h>
#include <sys/types.h>
#include <time.h>



using namespace std;

__device__ int mymin(int a, int b)
{
	int m = a;
	if(m > b)
		m=b;
	return m;
}
__device__ int min1(int a,int b,int c)
{
	int m=a;
    if(m>b)
		m=b;
    if(m>c)
		m=c;
    return m;
}	

int min2(int a,int b,int c)
{
	int m=a;
    if(m>b)
		m=b;
    if(m>c)
		m=c;
    return m;
}	

void swapDiagnolPointers(int **prev,int **current)
{
     int * temp = *prev;
     *prev      = *current;
     *current   = temp;
}

void swapDiagnolPointersForG(int **prevprev, int **prev,int **current)
{
     int * temp = *prevprev;
	 *prevprev = *prev;
     *prev      = *current;
     *current   = temp;
}

__global__ void less_than_n(int k,int numElements,int * D1,int * D2,int * I1,int * I2,int * G0,int *G1,int * G2,char *s1,char *s2, int gi, int ge)
		{
			int i =   blockIdx.x * blockDim.x + threadIdx.x;
			int s =0;
			
			if( i > numElements ) return ;
			
			if(i == 0)
				{
						G2[i]=gi+ge*k;
						I2[i]=G2[i]+ge;
				}
				if(i == k)
				{
						G2[i]=gi+ge*k;
						D2[i]=G2[i]+ge;
				}
				if( i>0 && i<k)
				{
					D2[i]=min(D1[i],G1[i]+gi)+ge;
					I2[i]=min(I1[i-1],G1[i-1]+gi)+ge;
					if(s1[i]!=s2[k-i])
						s=1;
					else
						s=0;
					G2[i]=min1(D2[i],I2[i],G0[i-1]+s);	
				}	
		}

__global__ void greater_than_n(int k,int numElements,int * D1,int * D2,int * I1,int * I2,int * G0,int *G1,int * G2,char *s1,char *s2,int n, int gi, int ge)
		{
			int i =   blockIdx.x * blockDim.x + threadIdx.x;
			int s =0;
			
			if( i > numElements ) return ;
	
				D2[i]=min(D1[i+1],G1[i+1]+gi)+ge;
				I2[i]=min(I1[i],G1[i]+gi)+ge;
				if(s1[i+(k-n)]!=s2[k-(i+k-n)])
					s=1;
				else
					s=0;
				if((k-n)==1)
					G2[i]=min1(D2[i],I2[i],G0[i]+s);
				else
					G2[i]=min1(D2[i],I2[i],G0[i+1]+s);
		}

int main(int argc, char** argv)
{
	int i,j,n,k,l;
	time_t t0, t1;
        clock_t c0,c1;
	char skip;

	scanf("%d",&n);
	printf("%d \n",n);
	while(1)
	{
		scanf("%c",&skip);
		if(skip == '\n')
			break;
	}
	size_t size = (n+1) * sizeof(char);
	size_t size1 = (n+1)*sizeof(int);

	char* h_s1 = (char*)malloc(size);
	char* h_s2 = (char*)malloc(size);
	
	for(i=1; i<=n; i++)
	{
		scanf("%c",&h_s1[i]);
	}
	while(1)
	{
		scanf("%c",&skip);
		if(skip == '\n')
			break;
	}
	for(i=1; i<=n; i++)
	{
		scanf("%c",&h_s2[i]);
	}
	

/*	t0=time(NULL);
        c0=clock();

        printf ("\tbegin (wall):            %ld\n", (long) t0);
        printf ("\tbegin (CPU):             %d\n", (int) c0);
*/	
	int* h_D0 = (int*)malloc(size1);
	int* h_D1 = (int*)malloc(size1);
	int* h_D2 = (int*)malloc(size1);
	int* h_G0 = (int*)malloc(size1);
	int* h_G1 = (int*)malloc(size1);
	int* h_G2 = (int*)malloc(size1);
	int* h_I0 = (int*)malloc(size1);
	int* h_I1 = (int*)malloc(size1);
	int* h_I2 = (int*)malloc(size1);
	
//	int D0[n+1],D1[n+1],D2[n+1],G0[n+1],G1[n+1],G2[n+1],I0[n+1],I1[n+1],I2[n+1];				//--- declaration of functions
	int gi,ge,s,cost;
	gi=2;ge=1;

		int *d_D0, *d_D1, *d_D2;
		hipMalloc(&d_D0, size1);
    	hipMalloc(&d_D1, size1);
		hipMalloc(&d_D2, size1);
    	int *d_G0, *d_G1, *d_G2;
		hipMalloc(&d_G0, size1);
    	hipMalloc(&d_G1, size1);
		hipMalloc(&d_G2, size1);
    	int *d_I0, *d_I1, *d_I2;
		hipMalloc(&d_I0, size1);
    	hipMalloc(&d_I1, size1);
		hipMalloc(&d_I2, size1);
	
	char *d_s1, *d_s2;
	hipMalloc(&d_s1, size);
	hipMalloc(&d_s2, size);

    hipMemcpy(d_s1, h_s1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_s2, h_s2, size, hipMemcpyHostToDevice);
	
		
	h_G1[0]=0;

	hipMemcpy(d_G1, h_G1, size1, hipMemcpyHostToDevice);
	int num_threads;
	int num_blocks;
	int numElements;
	
	for(k=1;k<=2*n;k++)
	{
		if(k <=n)
		{	
			 numElements = k+1;
			if( numElements <= 256) {
			 num_threads = numElements;
			 num_blocks =1 ;
			 }
			 else
			 {
			   num_threads = 256 ;
			   num_blocks = numElements/num_threads + ((numElements/num_threads == 0) ? 0 : 1) ;
			 }
			 
			 less_than_n<<<num_blocks,num_threads>>>(  k,  numElements,   d_D1, d_D2,  d_I1,d_I2,  d_G0, d_G1,  d_G2, d_s1, d_s2, gi, ge);
		}
		if(k > n)
		{
			numElements = n-(k-n)+1;
			if( numElements <= 256) {
			 num_threads = numElements;
			 num_blocks =1 ;
			 }
			 else
			 {
			   num_threads = 256 ;
			   num_blocks = numElements/num_threads + ((numElements/num_threads == 0) ? 0 : 1) ;
			 }
			 
			 greater_than_n<<<num_blocks,num_threads>>>(  k,  numElements,   d_D1, d_D2,  d_I1,d_I2,  d_G0, d_G1,  d_G2, d_s1, d_s2, n, gi, ge);
		}

		swapDiagnolPointers(&d_D1,&d_D2);
		swapDiagnolPointers(&d_I1,&d_I2);
		swapDiagnolPointersForG(&d_G0,&d_G1, &d_G2);
	}
	
	hipMemcpy(h_D2, d_D2, size1, hipMemcpyDeviceToHost);
	
    hipMemcpy(h_G2, d_G2, size1, hipMemcpyDeviceToHost);
   	
	hipMemcpy(h_I2, d_I2, size1, hipMemcpyDeviceToHost);

	hipMemcpy(h_D1, d_D1, size1, hipMemcpyDeviceToHost);
	
    hipMemcpy(h_G1, d_G1, size1, hipMemcpyDeviceToHost);
   	
	hipMemcpy(h_I1, d_I1, size1, hipMemcpyDeviceToHost);

	

	


	
	cost=min2(h_D1[0],h_I1[0],h_G1[0]);			//--- allignment cost
	
	cout<<"Optimal Allignment cost: "<<cost<<endl;
/*	
	t1=time(NULL);
        c1=clock();
        printf ("\telapsed wall clock time: %ld\n", (long) (t1 - t0));
        printf ("\telapsed CPU time:        %f\n", (float) (c1 - c0)/CLOCKS_PER_SEC);

	cudaFree(d_G0);
	cudaFree(d_G1);
	cudaFree(d_G2);
	cudaFree(d_D0);
	cudaFree(d_D1);
	cudaFree(d_D2);
	cudaFree(d_I0);
	cudaFree(d_I1);
	cudaFree(d_I2);
	cudaFree(d_s1);
	cudaFree(d_s2);
	
	free(h_s1);
	free(h_s2);
	free(h_G1);
	free(h_G2);
	free(h_D1);
	free(h_D2);
	free(h_I1);
	free(h_I2);
*/
	return 0;
}




