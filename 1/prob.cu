#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>


__global__ void copy_array(float* A, float* B)
{
//    int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

    B[i] = A[i];
}


__global__ void prefix_sum_extend(float* B, int t, int s)
{
//    int i = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i != 0)
	{	
		if(B[s+(2*i)-1] > B[s+(2*i)])
			B[t+i] = B[s+(2*i)];
		else
			B[t+i] = B[s+(2*i)-1];
	}
}

__global__ void prefix_sum_drop(float* B, float* C, int t, int s)
{
//    int i = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

	  if( i!= 0)
        {
            if(i==1)
                C[t+i]=B[t+i];
            else if(i%2==0)
                    C[t+i]=C[s+i/2];
                else
					if(C[s+((i-1)/2)] > B[t+i])
						C[t+i]=B[t+i];
					else
						C[t+i]=C[s+((i-1)/2)];
        }

}


int main(int argc, char** argv)
{
	int k;
    scanf("%d",&k);
    int i,m,t,s,h;
    int N = (int)pow(2.0,k);
    size_t size = (N+1) * sizeof(float);
    size_t size1 = (2*N)* sizeof(float);
    
    float* h_A = (float*)malloc(size);
    float* h_S = (float*)malloc(size);
    float* h_B = (float*)malloc(size1);
    float* h_C = (float*)malloc(size1);
    
    
    for(i=1;i<=N;i++)
        h_A[i]=rand()%10+1;
    
/*    for (int i=1; i<=N; i++)
         printf("%f ",h_A[i]);
    printf("\n");
*/
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_S;
    hipMalloc(&d_S, size);
    float* d_B;
    hipMalloc(&d_B, size1);
    float* d_C;
    hipMalloc(&d_C, size1);
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    int block_size, n_blocks; 
	if(N >=256)	
	{
		block_size = 256;
		n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
	}else{
		block_size =1;
		n_blocks = N+1;
	}
//    copy_array<<<1, N+1>>>(d_A, d_B);
  copy_array<<<n_blocks, block_size>>>(d_A, d_B);
     
 
    m = N;
    t=0;
    
    for(h=1;h<=k;h++)
    {
        s=t;
        t=t+m;
        m=m/2;
	if(m >=256)
        {
                block_size = 256;
                n_blocks = m/block_size + (m%block_size == 0 ? 0:1);
        }else{
                block_size =1;
                n_blocks = m+1;
        }

//        prefix_sum_extend<<<1, m+1>>>(d_B, t, s);
	 prefix_sum_extend<<<n_blocks, block_size>>>(d_B, t, s);


    }

    for(h=k;h>=0;h--)
    {
	if(m >=256)
        {
                block_size = 256;                                                      
                n_blocks = m/block_size + (m%block_size == 0 ? 0:1);                   
        }else{
                block_size =1;
                n_blocks = m+1;
        }

//        prefix_sum_drop<<<1, m+1>>>(d_B, d_C, t, s);
	prefix_sum_drop<<<n_blocks, block_size>>>(d_B, d_C, t, s);
        m=2*m;
        s=t;
        t=t-m;
    }
    
	if(N >=256)
        {
                block_size = 256;
                n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
        }else{
                block_size =1;
                n_blocks = N+1;
        }

//    copy_array<<<1, N+1>>>(d_C, d_S);
  copy_array<<<n_blocks, block_size>>>(d_C, d_S);  
    hipMemcpy(h_B, d_B, size1, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, size1, hipMemcpyDeviceToHost);
    hipMemcpy(h_S, d_S, size, hipMemcpyDeviceToHost);
    
	if(N < 256)
	{
    for (int i=1; i<=N; i++) 
        printf("%f ",h_S[i]);
    printf("\n");
	}
    hipFree(d_A);
    hipFree(d_S);
    hipFree(d_B);
    hipFree(d_C);
}


